#include "hip/hip_runtime.h"
/*
* Copyright (C) 2020-2025 MEmilio
*
* Authors: Khoa Nguyen
*
* Contact: Martin J. Kuehn <Martin.Kuehn@DLR.de>
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#ifdef MEMILIO_WITH_CUDA
#define R123_NO_CUDA_DEVICE_RANDOM 1
#endif

// Note: Place all includes after the defines
// #include "abm/household.h"
// #include "abm/lockdown_rules.h"
// #include "abm/model.h"
// #include "abm/common_abm_loggers.h"

#include <fstream>


#ifdef MEMILIO_WITH_CUDA
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// CUDA test functions remain unchanged...


// Define a larger test size for multi-core testing
#define CUDA_TEST_SIZE 1024

// Simple CUDA kernel to verify CUDA functionality
__global__ void testCudaKernel(int* result) 
{
    *result = 42;
}

// More complex kernel to test parallel computing on multiple cores
__global__ void testParallelKernel(float* input, float* output, int size) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Simple computation to verify parallel processing
        float value = input[idx];
        // Do some work to ensure the GPU is actually utilized
        for(int i = 0; i < 1000; i++) {
            value = sinf(value) * cosf(value) + sqrtf(fabs(value));
        }
        output[idx] = value;
    }
}

// Helper function to test if CUDA is working properly
bool testCuda()
{
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess || deviceCount == 0) {
        std::cout << "CUDA test: No CUDA devices found!" << std::endl;
        return false;
    }
    
    std::cout << "CUDA test: Found " << deviceCount << " CUDA device(s)" << std::endl;
    
    // Display information about the GPU
    hipDeviceProp_t deviceProp;
    for (int device = 0; device < deviceCount; device++) {
        hipGetDeviceProperties(&deviceProp, device);
        std::cout << "  Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "   Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "    Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "    Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
    }
    
    // Basic test - single value
    int* d_result;
    int h_result = 0;
    
    // Allocate device memory
    hipMalloc((void**)&d_result, sizeof(int));
    
    // Launch kernel
    testCudaKernel<<<1, 1>>>(d_result);
    
    // Copy result back
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_result);
    
    if (h_result != 42) {
        std::cout << "CUDA test: Failed! Basic kernel did not produce expected result." << std::endl;
        return false;
    }
    
    std::cout << "CUDA test: Basic single-thread test passed." << std::endl;
    
    // Extended test - multiple cores
    std::cout << "CUDA test: Running multi-core performance test..." << std::endl;
    
    // Create input data
    std::vector<float> h_input(CUDA_TEST_SIZE);
    std::vector<float> h_output(CUDA_TEST_SIZE);
    
    // Initialize input data
    for (int i = 0; i < CUDA_TEST_SIZE; i++) {
        h_input[i] = static_cast<float>(i) * 0.01f;
    }
    
    // Allocate device memory
    float* d_input;
    float* d_output;
    hipMalloc((void**)&d_input, CUDA_TEST_SIZE * sizeof(float));
    hipMalloc((void**)&d_output, CUDA_TEST_SIZE * sizeof(float));
    
    // Copy input data to device
    hipMemcpy(d_input, h_input.data(), CUDA_TEST_SIZE * sizeof(float), hipMemcpyHostToDevice);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record start time
    hipEventRecord(start);
    
    // Launch parallel kernel (use 256 threads per block)
    int blockSize = 256;
    int numBlocks = (CUDA_TEST_SIZE + blockSize - 1) / blockSize;
    testParallelKernel<<<numBlocks, blockSize>>>(d_input, d_output, CUDA_TEST_SIZE);
    
    // Record end time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy result back
    hipMemcpy(h_output.data(), d_output, CUDA_TEST_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    
    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Verify results (just check that they're not all zeros)
    bool hasNonZeroResults = false;
    for (int i = 0; i < CUDA_TEST_SIZE; i++) {
        if (h_output[i] != 0.0f) {
            hasNonZeroResults = true;
            break;
        }
    }
    
    if (!hasNonZeroResults) {
        std::cout << "CUDA test: Failed! Parallel kernel did not produce valid results." << std::endl;
        return false;
    }
    
    std::cout << "CUDA test: Multi-core test passed." << std::endl;
    std::cout << "CUDA test: Processing time: " << milliseconds << " ms" << std::endl;
    std::cout << "CUDA test: Success! CUDA is working properly with multiple cores." << std::endl;
    
    return true;
}
#endif

// Simple function to run ABM simulation that doesn't depend on CUDA
void runABMSimulation()
{
    // // This is a minimal example with children and adults < 60 year old.
    // // We divided them into 4 different age groups, which are defined as follows:
    // mio::set_log_level(mio::LogLevel::warn);
    // size_t num_age_groups         = 4;
    // const auto age_group_0_to_4   = mio::AgeGroup(0);
    // const auto age_group_5_to_14  = mio::AgeGroup(1);
    // const auto age_group_15_to_34 = mio::AgeGroup(2);
    // const auto age_group_35_to_59 = mio::AgeGroup(3);

    // // Create the model with 4 age groups.
    // auto model = mio::abm::Model(num_age_groups);

    // // Set same infection parameter for all age groups. For example, the incubation period is 4 days.
    // model.parameters.get<mio::abm::IncubationPeriod>() = 4.;

    // // Set the age group the can go to school is AgeGroup(1) (i.e. 5-14)
    // model.parameters.get<mio::abm::AgeGroupGotoSchool>()                    = false;
    // model.parameters.get<mio::abm::AgeGroupGotoSchool>()[age_group_5_to_14] = true;
    // // Set the age group the can go to work is AgeGroup(2) and AgeGroup(3) (i.e. 15-34 and 35-59)
    // model.parameters.get<mio::abm::AgeGroupGotoWork>().set_multiple({age_group_15_to_34, age_group_35_to_59}, true);

    // // Check if the parameters satisfy their contraints.
    // model.parameters.check_constraints();

    // // There are 10 households for each household group.
    // int n_households = 10;

    // // For more than 1 family households we need families. These are parents and children and randoms (which are distributed like the data we have for these households).
    // auto child = mio::abm::HouseholdMember(num_age_groups); // A child is 50/50% 0-4 or 5-14.
    // child.set_age_weight(age_group_0_to_4, 1);
    // child.set_age_weight(age_group_5_to_14, 1);

    // auto parent = mio::abm::HouseholdMember(num_age_groups); // A parent is 50/50% 15-34 or 35-59.
    // parent.set_age_weight(age_group_15_to_34, 1);
    // parent.set_age_weight(age_group_35_to_59, 1);

    // // Two-person household with one parent and one child.
    // auto twoPersonHousehold_group = mio::abm::HouseholdGroup();
    // auto twoPersonHousehold_full  = mio::abm::Household();
    // twoPersonHousehold_full.add_members(child, 1);
    // twoPersonHousehold_full.add_members(parent, 1);
    // twoPersonHousehold_group.add_households(twoPersonHousehold_full, n_households);
    // add_household_group_to_model(model, twoPersonHousehold_group);

    // // Three-person household with two parent and one child.
    // auto threePersonHousehold_group = mio::abm::HouseholdGroup();
    // auto threePersonHousehold_full  = mio::abm::Household();
    // threePersonHousehold_full.add_members(child, 1);
    // threePersonHousehold_full.add_members(parent, 2);
    // threePersonHousehold_group.add_households(threePersonHousehold_full, n_households);
    // add_household_group_to_model(model, threePersonHousehold_group);

    // // Add one social event with 5 maximum contacts.
    // // Maximum contacs limit the number of people that a person can infect while being at this location.
    // auto event = model.add_location(mio::abm::LocationType::SocialEvent);
    // model.get_location(event).get_infection_parameters().set<mio::abm::MaximumContacts>(5);
    // // Add hospital and ICU with 5 maximum contacs.
    // auto hospital = model.add_location(mio::abm::LocationType::Hospital);
    // model.get_location(hospital).get_infection_parameters().set<mio::abm::MaximumContacts>(5);
    // auto icu = model.add_location(mio::abm::LocationType::ICU);
    // model.get_location(icu).get_infection_parameters().set<mio::abm::MaximumContacts>(5);
    // // Add one supermarket, maximum constacts are assumed to be 20.
    // auto shop = model.add_location(mio::abm::LocationType::BasicsShop);
    // model.get_location(shop).get_infection_parameters().set<mio::abm::MaximumContacts>(20);
    // // At every school, the maximum contacts are 20.
    // auto school = model.add_location(mio::abm::LocationType::School);
    // model.get_location(school).get_infection_parameters().set<mio::abm::MaximumContacts>(20);
    // // At every workplace, maximum contacts are 20.
    // auto work = model.add_location(mio::abm::LocationType::Work);
    // model.get_location(work).get_infection_parameters().set<mio::abm::MaximumContacts>(20);

    // // Increase aerosol transmission for all locations
    // model.parameters.get<mio::abm::AerosolTransmissionRates>() = 10.0;
    // // Increase contact rate for all people between 15 and 34 (i.e. people meet more often in the same location)
    // model.get_location(work)
    //     .get_infection_parameters()
    //     .get<mio::abm::ContactRates>()[{age_group_15_to_34, age_group_15_to_34}] = 10.0;

    // // People can get tested at work (and do this with 0.5 probability) from time point 0 to day 10.
    // auto validity_period       = mio::abm::days(1);
    // auto probability           = 0.5;
    // auto start_date            = mio::abm::TimePoint(0);
    // auto end_date              = mio::abm::TimePoint(0) + mio::abm::days(10);
    // auto test_type             = mio::abm::TestType::Antigen;
    // auto test_parameters       = model.parameters.get<mio::abm::TestData>()[test_type];
    // auto testing_criteria_work = mio::abm::TestingCriteria();
    // auto testing_scheme_work   = mio::abm::TestingScheme(testing_criteria_work, validity_period, start_date, end_date,
    //                                                    test_parameters, probability);
    // model.get_testing_strategy().add_testing_scheme(mio::abm::LocationType::Work, testing_scheme_work);

    // // Assign infection state to each person.
    // // The infection states are chosen randomly with the following distribution
    // std::vector<double> infection_distribution{0.5, 0.3, 0.05, 0.05, 0.05, 0.05, 0.0, 0.0};
    // for (auto& person : model.get_persons()) {
    //     mio::abm::InfectionState infection_state = mio::abm::InfectionState(
    //         mio::DiscreteDistribution<size_t>::get_instance()(mio::thread_local_rng(), infection_distribution));
    //     auto rng = mio::abm::PersonalRandomNumberGenerator(person);
    //     if (infection_state != mio::abm::InfectionState::Susceptible) {
    //         person.add_new_infection(mio::abm::Infection(rng, mio::abm::VirusVariant::Wildtype, person.get_age(),
    //                                                      model.parameters, start_date, infection_state));
    //     }
    // }

    // // Assign locations to the people
    // for (auto& person : model.get_persons()) {
    //     const auto id = person.get_id();
    //     //assign shop and event
    //     model.assign_location(id, event);
    //     model.assign_location(id, shop);
    //     //assign hospital and ICU
    //     model.assign_location(id, hospital);
    //     model.assign_location(id, icu);
    //     //assign work/school to people depending on their age
    //     if (person.get_age() == age_group_5_to_14) {
    //         model.assign_location(id, school);
    //     }
    //     if (person.get_age() == age_group_15_to_34 || person.get_age() == age_group_35_to_59) {
    //         model.assign_location(id, work);
    //     }
    // }

    // // During the lockdown, social events are closed for 90% of people.
    // auto t_lockdown = mio::abm::TimePoint(0) + mio::abm::days(10);
    // mio::abm::close_social_events(t_lockdown, 0.9, model.parameters);

    // // Set start and end time for the simulation.
    // auto t0   = mio::abm::TimePoint(0);
    // auto tmax = t0 + mio::abm::days(5); // Reduced from 10 to 5 days for faster testing
    // auto sim  = mio::abm::Simulation(t0, std::move(model));

    // // Create a history object to store the time series of the infection states.
    // mio::History<mio::abm::TimeSeriesWriter, mio::abm::LogInfectionState> historyTimeSeries{
    //     Eigen::Index(mio::abm::InfectionState::Count)};

    // // Run the simulation until tmax with the history object.
    // std::cout << "Running ABM simulation..." << std::endl;
    // // sim.advance(tmax, historyTimeSeries);
    // std::cout << "ABM simulation completed." << std::endl;

    // // The results are written into the file "abm_minimal.txt" as a table
    // std::ofstream outfile("abm_minimal.txt");
    // std::get<0>(historyTimeSeries.get_log())
    //     .print_table({"S", "E", "I_NS", "I_Sy", "I_Sev", "I_Crit", "R", "D"}, 7, 4, outfile);
    std::cout << "Results written to abm_minimal.txt" << std::endl;
}

int main()
{
    // Test CUDA if enabled
    #ifdef MEMILIO_WITH_CUDA
    std::cout << "Testing CUDA capabilities..." << std::endl;
    
    // Guard the CUDA test with proper CUDA error handling
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed: " << hipGetErrorString(cudaStatus) << std::endl;
        std::cout << "CUDA test failed! Continuing without CUDA." << std::endl;
    }
    else {
        bool cudaWorking = testCuda();
        std::cout << "CUDA test " << (cudaWorking ? "passed!" : "failed!") << std::endl;
        
        // Reset the device before running non-CUDA code
        hipDeviceReset();
    }
    #else
    std::cout << "CUDA support is not enabled." << std::endl;
    #endif

    // Run ABM simulation (this doesn't use CUDA and shouldn't cause conflicts)
    // runABMSimulation();

    return 0;
}
