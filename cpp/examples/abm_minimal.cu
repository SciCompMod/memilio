/*
* Copyright (C) 2020-2025 MEmilio
*
* Authors: Khoa Nguyen
*
* Contact: Martin J. Kuehn <Martin.Kuehn@DLR.de>
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
#define MEMILIO_WITH_CUDA TRUE
#ifdef MEMILIO_WITH_CUDA
#define R123_NO_CUDA_DEVICE_RANDOM 1
#endif

// Note: Place all includes after the defines
// #include "abm/household.h"
// #include "abm/lockdown_rules.h"
// #include "abm/model.h"
// #include "abm/common_abm_loggers.h"

#include <iostream>


#ifdef MEMILIO_WITH_CUDA
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// CUDA test functions remain unchanged...


// Define a larger test size for multi-core testing
#define CUDA_TEST_SIZE 1024

// Simple CUDA kernel to verify CUDA functionality
__global__ void testCudaKernel(int* result) 
{
    *result = 42;
}

// More complex kernel to test parallel computing on multiple cores
__global__ void testParallelKernel(float* input, float* output, int size) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Simple computation to verify parallel processing
        float value = input[idx];
        // Do some work to ensure the GPU is actually utilized
        for(int i = 0; i < 1000; i++) {
            value = sinf(value) * cosf(value) + sqrtf(fabs(value));
        }
        output[idx] = value;
    }
}

// Helper function to test if CUDA is working properly
bool testCuda(int size)
{
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess || deviceCount == 0) {
        std::cout << "CUDA test: No CUDA devices found!" << std::endl;
        return false;
    }
    
    std::cout << "CUDA test: Found " << deviceCount << " CUDA device(s)" << std::endl;
    
    // Display information about the GPU
    hipDeviceProp_t deviceProp;
    for (int device = 0; device < deviceCount; device++) {
        hipGetDeviceProperties(&deviceProp, device);
        std::cout << "  Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "   Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "    Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "    Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
    }
    
    // Basic test - single value
    int* d_result;
    int h_result = 0;
    
    // Allocate device memory
    hipMalloc((void**)&d_result, sizeof(int));
    
    // Launch kernel
    testCudaKernel<<<1, size>>>(d_result);
    
    // Copy result back
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_result);
    
    if (h_result != 42) {
        std::cout << "CUDA test: Failed! Basic kernel did not produce expected result." << std::endl;
        return false;
    }
    
    std::cout << "CUDA test: Basic single-thread test passed." << std::endl;
    
    // Extended test - multiple cores
    std::cout << "CUDA test: Running multi-core performance test..." << std::endl;
    
    // Create input data
    std::vector<float> h_input(CUDA_TEST_SIZE);
    std::vector<float> h_output(CUDA_TEST_SIZE);
    
    // Initialize input data
    for (int i = 0; i < CUDA_TEST_SIZE; i++) {
        h_input[i] = static_cast<float>(i) * 0.01f;
    }
    
    // Allocate device memory
    float* d_input;
    float* d_output;
    hipMalloc((void**)&d_input, CUDA_TEST_SIZE * sizeof(float));
    hipMalloc((void**)&d_output, CUDA_TEST_SIZE * sizeof(float));
    
    // Copy input data to device
    hipMemcpy(d_input, h_input.data(), CUDA_TEST_SIZE * sizeof(float), hipMemcpyHostToDevice);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record start time
    hipEventRecord(start);
    
    // Launch parallel kernel (use 256 threads per block)
    int blockSize = 256;
    int numBlocks = (CUDA_TEST_SIZE + blockSize - 1) / blockSize;
    testParallelKernel<<<numBlocks, blockSize>>>(d_input, d_output, CUDA_TEST_SIZE);
    
    // Record end time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy result back
    hipMemcpy(h_output.data(), d_output, CUDA_TEST_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    
    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Verify results (just check that they're not all zeros)
    bool hasNonZeroResults = false;
    for (int i = 0; i < CUDA_TEST_SIZE; i++) {
        if (h_output[i] != 0.0f) {
            hasNonZeroResults = true;
            break;
        }
    }
    
    if (!hasNonZeroResults) {
        std::cout << "CUDA test: Failed! Parallel kernel did not produce valid results." << std::endl;
        return false;
    }
    
    std::cout << "CUDA test: Multi-core test passed." << std::endl;
    std::cout << "CUDA test: Processing time: " << milliseconds << " ms" << std::endl;
    std::cout << "CUDA test: Success! CUDA is working properly with multiple cores." << std::endl;
    
    return true;
}
#endif


int abm_minimal_main(int size)
{
    // Test CUDA if enabled
    #ifdef MEMILIO_WITH_CUDA
    std::cout << "Testing CUDA capabilities..." << std::endl;
    
    // Guard the CUDA test with proper CUDA error handling
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed: " << hipGetErrorString(cudaStatus) << std::endl;
        std::cout << "CUDA test failed! Continuing without CUDA." << std::endl;
    }
    else {
        bool cudaWorking = testCuda(size);
        std::cout << "CUDA test " << (cudaWorking ? "passed!" : "failed!") << std::endl;
        
        // Reset the device before running non-CUDA code
        hipDeviceReset();
    }
    #else
    std::cout << "CUDA support is not enabled." << std::endl;
    #endif

    // Run ABM simulation (this doesn't use CUDA and shouldn't cause conflicts)
    // runABMSimulation();

    return 0;
}
