#include "hip/hip_runtime.h"
#define R123_NO_CUDA_DEVICE_RANDOM 1
// #define CUDA_TEST_SIZE 1024
#include "boost/predef/compiler/nvcc.h"
#include <hip/hip_runtime.h>

#include "memilio/math/integrator.h"
#include "memilio/math/adapt_rk.h"
#include "memilio/math/stepper_wrapper.h"

#include <iostream>
#include <memory>
#include <cmath>

uint64_t seed                  = 147634;
// Numerical Recipes, ranqd1
const uint64_t rand_modulus    = (uint64_t(1) << 32);
const uint64_t rand_multiplier = 1664525;
const uint64_t rand_increment  = 1013904223;

uint64_t randc() {
    seed = (rand_multiplier * seed + rand_increment) & (rand_modulus - 1);
    return seed; 
}

double uniform_rand(double min, double max) {
    auto val = randc() / ((double)rand_modulus) * (max - min) + min;
    mio::log_info("rand {}", val);
    return val;
}

Eigen::VectorXd t_offset, t_scale;
// Eigen::VectorXd amplitude;
Eigen::MatrixXd amplitude_lincomb;

void set_params(Eigen::Index size, Eigen::Index band_width, double min, double max) {
    t_offset = Eigen::VectorXd::Zero(size);
    t_scale =  Eigen::VectorXd::Zero(size);
    // amplitude = Eigen::VectorXd(size);
    amplitude_lincomb = Eigen::MatrixXd::Zero(size, size);


    for (Eigen::Index i = 0; i < size; i++) {
        t_offset[i] = uniform_rand(min, max);
        t_scale[i] = uniform_rand(min, max);
        // amplitude[i] = uniform_rand(min, max);
        
        for (Eigen::Index j = i - (band_width / 2); j < i + ((band_width + 1) / 2) && j < size; j++) {
            if (j >= 0)
                amplitude_lincomb(i,j) = uniform_rand(min, max);
        } 
    }
}


void rhs(Eigen::Ref<const Eigen::VectorXd> x, double t, Eigen::Ref<Eigen::VectorXd> dxdt) {
    // dxdt = amplitude.array() * (t * t_scale + t_offset).array().sin();
    dxdt = amplitude_lincomb.matrix() * (t * t_scale + t_offset).array().sin().matrix();
}

int main()
{   
    using namespace mio;
    set_log_level(LogLevel::off);
    
    mio::log_debug("Enter Main");
    
    const int size = 3000;

    // Guard the CUDA test with proper CUDA error handling
    // hipError_t cudaStatus = hipSetDevice(0);
    // if (cudaStatus != hipSuccess) {
    //     std::cerr << "CUDA initialization failed: " << hipGetErrorString(cudaStatus) << std::endl;
    //     std::cout << "CUDA test failed! Continuing without CUDA." << std::endl;
    // }
    // else {
    //     std::cout << "CUDA initialization succeeded." <<  std::endl;
        
    //     hipDeviceReset();
    // } 

    // TODO: nvidia-x-markers??

    set_params(size, 7, -3.0, 3.0);
    mio::log_debug("Params Set");

    // std::cout << amplitude_lincomb << "\n";

    const double abs_tol = 1e-3, rel_tol = 1e-8, min_dt = 1e-2, max_dt = 1e+2;
    // auto core = std::make_shared<mio::ControlledStepperWrapper<double, boost::numeric::odeint::runge_kutta_cash_karp54>>(abs_tol, rel_tol, min_dt, max_dt);
    auto core = std::make_shared<mio::RKIntegratorCore<double>>(abs_tol, rel_tol, min_dt, max_dt);

    mio::log_debug("Core Set");

    OdeIntegrator<double> integrator(core);

    mio::log_debug("Integrator Set");

    TimeSeries<double> results(0, Eigen::VectorXd::Zero(size));

    const double tmax = 100 * M_PI;
    double dt = 0.1;

    mio::log_debug("Results Set");
    mio::log_debug("Integrating...");

    integrator.advance(rhs, tmax, dt, results);
    
    mio::log_debug("Integration Finished");

    if (size < 5)
        results.print_table();
    else
        std::cout << "Num time steps: " << results.get_num_time_points() << "\n"; 

    mio::log_debug("Exit Main");
    return 0;
}
