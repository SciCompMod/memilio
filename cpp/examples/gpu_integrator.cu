#include "hip/hip_runtime.h"
#define R123_NO_CUDA_DEVICE_RANDOM 1
// #define CUDA_TEST_SIZE 1024
#include "boost/predef/compiler/nvcc.h"
#include <hip/hip_runtime.h>

#include "memilio/math/integrator.h"
#include "memilio/math/adapt_rk.h"

#include <iostream>
#include <memory>
#include <cmath>

Eigen::VectorX<double> t_offset, t_scale, amplitude;

void set_params(Eigen::Index size) {
    t_offset = Eigen::VectorX<double>::Random(size);
    t_scale = Eigen::VectorX<double>::Random(size);
    amplitude = Eigen::VectorX<double>::Random(size);
}


void rhs(Eigen::Ref<const Eigen::VectorX<double>> x, double t, Eigen::Ref<Eigen::VectorX<double>> dxdt) {
    dxdt = amplitude.array() * (t * t_scale + t_offset).array().sin();
}

int main()
{   
    using namespace mio;
    
    const int size = 10;

    // Guard the CUDA test with proper CUDA error handling
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed: " << hipGetErrorString(cudaStatus) << std::endl;
        std::cout << "CUDA test failed! Continuing without CUDA." << std::endl;
    }
    else {
        std::cout << "CUDA initialization succeeded." <<  std::endl;
        
        hipDeviceReset();
    } 

    // TODO: nvidia-x-markers??

    set_params(size);

    auto core = std::make_shared<RKIntegratorCore<double>>(1e-3, 1e-8, 0.001, 1.);
    OdeIntegrator<double> integrator(core);

    Eigen::VectorX<double> init(size);
    init.fill(1.0); 
    TimeSeries<double> results(0, init);
    results.print_table();

    const double tmax = 6.29;
    double dt = 0.1;

    integrator.advance(rhs, tmax, dt, results);

    results.print_table();

    return 0;
}
