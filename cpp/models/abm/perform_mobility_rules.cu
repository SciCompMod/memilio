#include "hip/hip_runtime.h"
/* 
* Copyright (C) 2020-2025 MEmilio
*
* Authors: Julia Bicker, Sascha Heinz Korf, Carlotta von Gerstein
*
* Contact: Martin J. Kuehn <Martin.Kuehn@DLR.de>
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <vector>
#include <hip/hip_runtime.h>
#include "abm/interface_cuda.h"
#include "abm/location_type.h"
#include "abm/infection_state.h"
#include "abm/time.h"
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <chrono>  // Add this for timing measurements

namespace mio {
namespace abm {

__constant__ int weekend_cutoff = 5 ;
__constant__ int event_gotimeweekend = 10 ;
__constant__ int event_gotime_weekday = 19 ;
__constant__ int event_comebacktime = 20 ;


__device__ LocationType get_buried(const GPurson& person, int t){
    auto current_loc = person.current_loc;
    if (person.infection_state == InfectionState::Dead) {
        return LocationType::Cemetery;
    }
    return current_loc;
}

__device__ LocationType return_home_when_recovered(const GPurson& person, int t){
    auto current_loc = person.current_loc;
    if ((current_loc == LocationType::Hospital || current_loc == LocationType::ICU) &&
        person.infection_state == InfectionState::Recovered) {
        return LocationType::Home;
    }
    return current_loc;
}

__device__ LocationType go_to_hospital(const GPurson& person, int t){
    auto current_loc = person.current_loc;
    if (person.infection_state == InfectionState::InfectedSevere) {
        return LocationType::Hospital;
    }
    return current_loc;
}

__device__ LocationType go_to_icu(const GPurson& person, int t){
    auto current_loc = person.current_loc;
    if (person.infection_state == InfectionState::InfectedCritical) {
        return LocationType::ICU;
    }
    return current_loc;
}

__device__ bool random_transition(hiprandState_t* rng_state, double dt_days, double rate){
    float u_exp = hiprand_uniform(rng_state);
    double v = -std::logf(u_exp) / rate;
    return v < dt_days;
}

__device__ LocationType go_to_event(const GPurson& person, hiprandState_t* rng_state, int t, double dt_days, double rate){
    auto current_loc = person.current_loc;
    if(current_loc == LocationType::Home){
        if(random_transition(rng_state, dt_days, rate)){
            return LocationType::SocialEvent;
        }
        // if(t%24 >= event_gotime_weekday){
        //     return LocationType::SocialEvent;
        // }
    }
    else if(current_loc == LocationType::SocialEvent){
        if(t%24 >= event_comebacktime && person.time_at_location_hours >= 2.0){
            return LocationType::Home;
        }
    }
    return current_loc;
}
    
__device__ LocationType try_mobility_rule(const GPurson& person, hiprandState_t* rng_state, int t, double dt_days, double rate){
    auto loc_type = get_buried(person,t);
    if(loc_type != person.current_loc){
        return loc_type;
    }
    loc_type = return_home_when_recovered(person,t);
    if(loc_type != person.current_loc){
        return loc_type;
    }
    loc_type = go_to_hospital(person,t);
    if(loc_type != person.current_loc){
        return loc_type;
    }
    loc_type = go_to_icu(person,t);
    if(loc_type != person.current_loc){
        return loc_type;
    }
    loc_type = go_to_event(person, rng_state, t, dt_days, rate);
    if(loc_type != person.current_loc){
        return loc_type;
    }
    return person.current_loc;
}

// CUDA kernel for mobility rule get_buried
__global__ void next_loc(const GPurson* persons, LocationType* results, int num_persons, int t, double dt_days, unsigned long long seed, hiprandState_t* states, double rate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_persons) {
        hiprand_init(seed, idx, 0 &states[idx]);
        results[persons[idx].id] = try_mobility_rule(persons[idx], states[idx], t, dt_days, rate);
    }
}


// Helper function to measure elapsed time
double elapsedMilliseconds(const std::chrono::high_resolution_clock::time_point& start) {
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration<double, std::milli>(end - start).count();
}

// CUDA implementation for LogTimeAtLocationForEachPerson
std::vector<LocationType> mobility_rules(const std::vector<GPurson>& gPursons, int num_persons, int t, double dt_days, double rate, unsigned long long seed) 
{
    // Start timing
    auto start_total = std::chrono::high_resolution_clock::now();
    
    // Create results vector
    std::vector<LocationType> next_locs(num_persons, LocationType(0));
    
    if (num_persons == 0 || gPursons.empty()) {
        return next_locs;
    }
    
    printf("CUDA Performance Report:\n");
    printf("Number of persons: %d\n", num_persons);
    
    // Get device properties to verify CUDA is working
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using GPU: %s with %d multiprocessors\n", prop.name, prop.multiProcessorCount);
    
    // Start memory allocation timing
    auto start_alloc = std::chrono::high_resolution_clock::now();
    
    // Allocate device memory
    GPurson* d_persons = nullptr;
    LocationType* d_results = nullptr;
    hiprandState_t* d_rng_states = nullptr;
    
    hipError_t err = hipMalloc(&d_persons, num_persons * sizeof(GPurson));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMalloc persons): %s\n", hipGetErrorString(err));
        return next_locs;
    }
    
    err = hipMalloc(&d_results, num_persons * sizeof(LocationType));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMalloc results): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        return next_locs;
    }

    err = hipMalloc(&d_rng_states, num_persons * sizeof(hiprandState_t));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMalloc rng_states): %s\n", hipGetErrorString(err));
        hipFree(d_rng_states);
        return next_locs;
    }
    
    // Initialize results to zero
    err = hipMemset(d_results, 0, num_persons * sizeof(LocationType));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMemset): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        hipFree(d_results);
        return next_locs;
    }
    
    printf("Memory allocation time: %.3f ms\n", elapsedMilliseconds(start_alloc));
    
    // Start memory copy timing
    auto start_copy = std::chrono::high_resolution_clock::now();
    
    // Copy data to device
    err = hipMemcpy(d_persons, gPursons.data(), gPursons.size() * sizeof(GPurson), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMemcpy to device): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        hipFree(d_results);
        return next_locs;
    }
    
    printf("H2D memory copy time: %.3f ms\n", elapsedMilliseconds(start_copy));
    
    // Start kernel timing
    auto start_kernel = std::chrono::high_resolution_clock::now();
    
    // Launch kernel
    int blockSize = 256;
    int numBlocks = (num_persons + blockSize - 1) / blockSize;
    printf("Launching kernel with %d blocks of %d threads\n", numBlocks, blockSize);
    
    next_loc<<<numBlocks, blockSize>>>(d_persons, d_results, num_persons, t, dt_days, seed, d_rng_states, rate);
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (kernel launch): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        hipFree(d_results);
        return next_locs;
    }
    
    // Wait for kernel to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (device sync): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        hipFree(d_results);
        return next_locs;
    }
    
    printf("Kernel execution time: %.3f ms\n", elapsedMilliseconds(start_kernel));
    
    // Start copy back timing
    auto start_copyback = std::chrono::high_resolution_clock::now();
    
    // Copy results back to host
    err = hipMemcpy(next_locs.data(), d_results, num_persons * sizeof(LocationType), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMemcpy to host): %s\n", hipGetErrorString(err));
    }
    
    printf("D2H memory copy time: %.3f ms\n", elapsedMilliseconds(start_copyback));
    
    // Start cleanup timing
    auto start_cleanup = std::chrono::high_resolution_clock::now();
    
    // Free device memory
    hipFree(d_persons);
    hipFree(d_results);
    
    printf("Cleanup time: %.3f ms\n", elapsedMilliseconds(start_cleanup));
    printf("Total CUDA time: %.3f ms\n", elapsedMilliseconds(start_total));
    
    // // For comparison, let's also measure the CPU version
    // auto start_cpu = std::chrono::high_resolution_clock::now();
    // std::vector<int> cpu_results(num_persons, 0.0);
    
    // // Sequential CPU version
    // for (const auto& person : gPursons) {
    //     cpu_results[person.id] = person.time_at_location_hours;
    // }
    
    // printf("CPU sequential time: %.3f ms\n", elapsedMilliseconds(start_cpu));
    // printf("Speedup: %.2fx\n", elapsedMilliseconds(start_cpu) / elapsedMilliseconds(start_total));
    
    // // Verify results match between CPU and GPU
    // bool results_match = true;
    // for (int i = 0; i < num_persons; i++) {
    //     if (std::abs(tal[i] - cpu_results[i]) > 1e-6) {
    //         results_match = false;
    //         break;
    //     }
    // }
    // printf("Results match between CPU and GPU: %s\n", results_match ? "YES" : "NO");
    
    return next_locs;
}

}} // namespace mio::abm


