#include "hip/hip_runtime.h"
/* 
* Copyright (C) 2020-2025 MEmilio
*
* Authors: Sascha Korf
*
* Contact: Martin J. Kuehn <Martin.Kuehn@DLR.de>
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <vector>
#include <hip/hip_runtime.h>
#include "abm/interface_cuda.h"
#include <stdio.h>
#include <chrono>  // Add this for timing measurements

namespace mio {
namespace abm {

// CUDA kernel to compute time at location for each person
__global__ void computeTimeAtLocationKernel(const CudaPerson* persons, double* results, int num_persons) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_persons) {
        results[persons[idx].id] = persons[idx].time_at_location_hours;
    }
}

// Helper function to measure elapsed time
double elapsedMilliseconds(const std::chrono::high_resolution_clock::time_point& start) {
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration<double, std::milli>(end - start).count();
}

// CUDA implementation for LogTimeAtLocationForEachPerson
std::vector<double> logTimeAtLocationCuda(const std::vector<CudaPerson>& cuda_persons, int num_persons) 
{
    // Start timing
    auto start_total = std::chrono::high_resolution_clock::now();
    
    // Create results vector
    std::vector<double> tal(num_persons, 0.0);
    
    if (num_persons == 0 || cuda_persons.empty()) {
        return tal;
    }
    
    printf("CUDA Performance Report:\n");
    printf("Number of persons: %d\n", num_persons);
    
    // Get device properties to verify CUDA is working
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using GPU: %s with %d multiprocessors\n", prop.name, prop.multiProcessorCount);
    
    // Start memory allocation timing
    auto start_alloc = std::chrono::high_resolution_clock::now();
    
    // Allocate device memory
    CudaPerson* d_persons = nullptr;
    double* d_results = nullptr;
    
    hipError_t err = hipMalloc(&d_persons, num_persons * sizeof(CudaPerson));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMalloc persons): %s\n", hipGetErrorString(err));
        return tal;
    }
    
    err = hipMalloc(&d_results, num_persons * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMalloc results): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        return tal;
    }
    
    // Initialize results to zero
    err = hipMemset(d_results, 0, num_persons * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMemset): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        hipFree(d_results);
        return tal;
    }
    
    printf("Memory allocation time: %.3f ms\n", elapsedMilliseconds(start_alloc));
    
    // Start memory copy timing
    auto start_copy = std::chrono::high_resolution_clock::now();
    
    // Copy data to device
    err = hipMemcpy(d_persons, cuda_persons.data(), cuda_persons.size() * sizeof(CudaPerson), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMemcpy to device): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        hipFree(d_results);
        return tal;
    }
    
    printf("H2D memory copy time: %.3f ms\n", elapsedMilliseconds(start_copy));
    
    // Start kernel timing
    auto start_kernel = std::chrono::high_resolution_clock::now();
    
    // Launch kernel
    int blockSize = 256;
    int numBlocks = (num_persons + blockSize - 1) / blockSize;
    printf("Launching kernel with %d blocks of %d threads\n", numBlocks, blockSize);
    
    computeTimeAtLocationKernel<<<numBlocks, blockSize>>>(d_persons, d_results, num_persons);
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (kernel launch): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        hipFree(d_results);
        return tal;
    }
    
    // Wait for kernel to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (device sync): %s\n", hipGetErrorString(err));
        hipFree(d_persons);
        hipFree(d_results);
        return tal;
    }
    
    printf("Kernel execution time: %.3f ms\n", elapsedMilliseconds(start_kernel));
    
    // Start copy back timing
    auto start_copyback = std::chrono::high_resolution_clock::now();
    
    // Copy results back to host
    err = hipMemcpy(tal.data(), d_results, num_persons * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (hipMemcpy to host): %s\n", hipGetErrorString(err));
    }
    
    printf("D2H memory copy time: %.3f ms\n", elapsedMilliseconds(start_copyback));
    
    // Start cleanup timing
    auto start_cleanup = std::chrono::high_resolution_clock::now();
    
    // Free device memory
    hipFree(d_persons);
    hipFree(d_results);
    
    printf("Cleanup time: %.3f ms\n", elapsedMilliseconds(start_cleanup));
    printf("Total CUDA time: %.3f ms\n", elapsedMilliseconds(start_total));
    
    // For comparison, let's also measure the CPU version
    auto start_cpu = std::chrono::high_resolution_clock::now();
    std::vector<double> cpu_results(num_persons, 0.0);
    
    // Sequential CPU version
    for (const auto& person : cuda_persons) {
        cpu_results[person.id] = person.time_at_location_hours;
    }
    
    printf("CPU sequential time: %.3f ms\n", elapsedMilliseconds(start_cpu));
    printf("Speedup: %.2fx\n", elapsedMilliseconds(start_cpu) / elapsedMilliseconds(start_total));
    
    // Verify results match between CPU and GPU
    bool results_match = true;
    for (int i = 0; i < num_persons; i++) {
        if (std::abs(tal[i] - cpu_results[i]) > 1e-6) {
            results_match = false;
            break;
        }
    }
    printf("Results match between CPU and GPU: %s\n", results_match ? "YES" : "NO");
    
    return tal;
}

}} // namespace mio::abm

